#include "hip/hip_runtime.h"
#include <math.h>
#include "ActivationFunction.hpp"

double ActivationFunction::activate(FunctionType t_type, double t_input) {
	switch (t_type)
	{
	case ActivationFunction::FunctionType::Identity:
		return t_input;
	case ActivationFunction::FunctionType::TanH:
		return tanh(t_input);
	default:
		return nan("");
	}
}

ActivationFunction::FunctionType ActivationFunction::getFromRandom(float t_randomNumber) {
	if (t_randomNumber > 0.5)
		return ActivationFunction::FunctionType::Identity;
	else
		return ActivationFunction::FunctionType::TanH;
}